/*
 * HzUpdaterTE.cpp
 *
 *  Created on: 29 янв. 2016 г.
 *      Author: aleksandr
 */

#include "HzUpdaterTE.h"
#include "SmartIndex.h"
//   o o o o x
//   o o o o x
//   o o o o x
//   o o o o x
//   x x x x x

__host__ __device__
void HzUpdaterTE::operator() (const int indx) {
	// m и n - индексы в полноценных массивах
	// sizeY - размер полноценнго массива
	int m = indx/(sizeY);
	int n = indx%(sizeY);
	float Chze = S / 377.0;
	if ( (m<sizeX-1) && (n<sizeY-1) ) {
		Hz(m, n) = Hz(m, n) - Chze*((Ex(m, n + 1) - Ex(m, n)) - (Ey(m + 1, n) - Ey(m, n)));
	}
}
