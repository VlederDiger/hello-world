#include "hip/hip_runtime.h"
/*
 * UpdaterEz1D.cpp
 *
 *  Created on: 25 янв. 2016 г.
 *      Author: aleksandr
 */

#include "UpdaterIntensityTM.h"

__device__
void UpdaterIntensityTM::operator() (const int indx) {
	#define Ex(M, N) Ex[(M) * (sizeY) + (N)]
    #define Ey(M, N) Ey[(M) * (sizeY-1) + (N)]
	const int n = indx % sizeY;
	const int m = indx / sizeY;
	intensity[indx] = intensity[indx] + sqrt(Ex(firstX + m*stepX, firstY + n*stepX)*Ex(firstX + m*stepX, firstY + n*stepX)+Ey(firstX + m*stepX, firstY + n*stepX)*Ey(firstX + m*stepX, firstY + n*stepX));


}
